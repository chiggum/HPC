#include "hip/hip_runtime.h"
/*
COMPILE WITH
nvcc fractal.cu -lopencv_highgui -lopencv_core -lopencv_imgproc -o fractal
*/
#include <iostream>
#include <cstdlib>
#include <hip/hip_complex.h>
#include "opencv2/core/core.hpp"
#include "opencv2/gpu/gpu.hpp"
#include "opencv2/highgui/highgui.hpp"

#define cudaMemcpyHTD(dest, src, nBytes) hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice)
#define cudaMemcpyDTH(dest, src, nBytes) hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost)

#define BLOCKSIZE 512


/**********************************************
Configuration:
***********************************************/
double interceptRealMin = -1;
double interceptRealMax = 1;
double interceptImgMin = -1;
double interceptImgMax = 1;
double realOffset = 0.01;
double imgOffset = 0.01;
double maxZAbs = 10;
int maxN = 255;
int minN = 1;
int decayN = 1;
double offset = 4;
double realMin = -2;
double imgMin = -2;	
double offMul = 1;
/*********************************************
*********************************************/
	
	

__device__ __forceinline__ hipDoubleComplex my_cexpf(hipDoubleComplex z) {
    hipFloatComplex res;
    float t = expf (z.x);
    sincosf(z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return hipComplexFloatToDouble(res);
}

__device__ hipDoubleComplex getFuncVal(hipDoubleComplex z, hipDoubleComplex c, int categ) {
	switch(categ) {
		case 1:
			return hipCadd(hipCmul(z,z), c);
		case 2:
			return hipCadd(hipCmul(hipCmul(z, z),z), c);
		case 3:
			return hipCadd(hipCmul(hipCmul(hipCmul(z, z), z),z), c);
		case 4:
			return hipCadd(hipCmul(hipCmul(hipCmul(hipCmul(z, z), z), z),z), c);
		case 5:
			return hipCadd(my_cexpf(z), c);
		case 6:
			return hipCadd(my_cexpf(hipCmul(hipCmul(hipCmul(z, z), z),z)), c);
		case 7:
			return hipCadd(hipCmul(my_cexpf(z), z),z);
		case 8:
			return hipCadd(hipCmul(hipCmul(my_cexpf(z), z), z),z);
		case 9:
			return hipCadd(hipCmul(hipCmul(hipCmul(my_cexpf(z), z), z), z),z);
		default:
			return hipCadd(hipCmul(z,z), c);
	}
}

__global__ void fractalForm(int *mat, int maxZAbs, int maxN, int minN, int decayN, double iReal,
							double iImg, int categ, double rMin, double rMax, double iMin, double iMax,
							int H, int W) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx >= H*W)
		return;
	int i_ = idx/W;
	int j_ = idx%W;
	double re = rMin + (i_*(rMax-rMin))/(1.0*H);
	double im = iMin + (j_*(iMax-iMin))/(1.0*W);
	hipDoubleComplex z = make_hipDoubleComplex(re, im);
	hipDoubleComplex c = make_hipDoubleComplex(iReal, iImg);
	size_t n;
	for(n = maxN; n >= minN && hipCabs(z) < maxZAbs; n-=decayN) {
		z = getFuncVal(z, c, categ);
	}
	
	mat[j_ + i_*W] = n;
}

int main(int argc, char **argv) {
	
	if(argc != 3) {
		std::cout << "Usage: " << argv[0] << " " << "height=width" << " " << "category in 1 to 10" << std::endl;
		exit(EXIT_FAILURE);
	} else {
		std::cout << "press\nq/z for quit\nf,g\nv,b\no,p\nk,l\nn,m\nu,i" << std::endl;
	}
	int H = atoi(argv[1]);
	int W = H;
	int category = atoi(argv[2]);
	
	
	int *h_mat, *d_mat;
	h_mat = new int[H*W];
	hipMalloc((void**)&d_mat, H*W*sizeof(int));
	
	dim3 threadsPerBlock(BLOCKSIZE);
	dim3 numBlocks((H*W-1)/threadsPerBlock.x + 1);
	
	cv::Mat finalImg(H, W, CV_8UC3);
	
	for(double iReal = (interceptRealMin+interceptRealMax)/2, iImg = (interceptImgMin+interceptImgMax)/2;;) {
		fractalForm<<<numBlocks, threadsPerBlock>>>(d_mat, maxZAbs, maxN, minN, decayN, iReal,
													iImg, category, realMin, realMin + offset, 
													imgMin, imgMin + offset, H, W);
	
		cudaMemcpyDTH(h_mat, d_mat, H*W*sizeof(int));

		for(size_t i = 0; i < H; ++i) {
		     for(size_t j = 0; j < W; ++j) {
				//finalImg.at<uchar>(i,j) = h_mat[j+i*W];
				finalImg.at<cv::Vec3b>(i, j)[2] = h_mat[j+i*W];
		     }
		}
		cv::imshow("fractal", finalImg);
		char c = cv::waitKey(0);
		if(c == 'f') {
			iReal-=realOffset;
		} else if(c == 'g') {
			iReal+=realOffset;
		} else if(c == 'v') {
			iImg -= imgOffset;
		} else if(c == 'b') {
			iImg += imgOffset;
		} else if(c == 'z' || c == 'q') {
			break;
		} else if(c == 's') {
			cv::imwrite("fractal.png", finalImg);
		} else if(c == 'o') {
			offset -= 0.05*offMul;
		} else if(c == 'p') {
			offset += 0.05*offMul;
		} else if(c == 'k') {
			realMin -= 0.05*offMul;
		} else if(c == 'l') {
			realMin += 0.05*offMul;
		} else if(c == 'n') {
			imgMin -= 0.05*offMul;
		} else if(c == 'm') {
			imgMin += 0.05*offMul;
		} else if(c == 'u') {
			offMul/=10;
		} else if(c == 'i') {
			offMul*=10;
		}
		
	}
	return 0;
}